/*
* CUDA training code for my neural network!
* Variable declarations starting with gpu are allocated in gpu memory
*/

#include <iostream>

#include "hip/hip_runtime.h"
#include ""

#include "DenseLayer.h"
#include "InputLayer.h"
#include "OutputLayer.h"
#include "LSTMLayer.h"
#include "BaseLayer.h"
//#define DEBUG

#define CUDA_CHECK(err, code) \
    if (err != hipSuccess) { \
        printf("\nCUDA: Error (%s): %s at line %d\n", code, hipGetErrorString(err), __LINE__); \
        exit(EXIT_FAILURE);\
    }

const int threadsPerBlock = 64;

BaseLayer** gpu_allLayer = nullptr;
BaseLayer** cpu_allLayer = nullptr;
int allLayerCount;

float * gpu_desiredValues;

void PrintLayerInfo(BaseLayer * layer, const char* layerName) {
    printf("%s - Weights: %p, Biases: %p, NeuronValues: %p, Errors: %p, Size: %d\n",
        layerName, layer->Weights, layer->Biases, layer->NeuronValues, layer->Errors, layer->Size);
}

void FeedForward() {
    hipError_t err;

    //first layer can be skipped -> feed forward all hidden and output:
    for (int i = 1; i < allLayerCount; i++) {
        gpu_allLayer[i]->FeedForward(threadsPerBlock);
    }

    err = hipDeviceSynchronize();
    CUDA_CHECK(err, "Feed Forward Synchronize Threads");
}

extern "C" __declspec(dllexport) void Train(float* inputs, float* desired, int size, float learningRate) {
    hipError_t err;

    //copy the next inputs & outputs to the gpu memory
    err = hipMemcpy(gpu_allLayer[0]->NeuronValues, inputs, cpu_allLayer[0]->Size * sizeof(float), hipMemcpyHostToDevice);
    CUDA_CHECK(err, "Memcpy Inputs");
    
    if (gpu_desiredValues == nullptr) {
        err = hipMalloc(&gpu_desiredValues, cpu_allLayer[allLayerCount - 1]->Size * sizeof(float));
        CUDA_CHECK(err, "Malloc Desired Values");
    }
    err = hipMemcpy(gpu_desiredValues, desired, cpu_allLayer[allLayerCount - 1]->Size * sizeof(float), hipMemcpyHostToDevice);
    CUDA_CHECK(err, "Memcopy Desired Values");

    FeedForward();
    
    for (int i = allLayerCount - 1; i >= 0; i--) {
        gpu_allLayer[i]->Train(threadsPerBlock, gpu_desiredValues, learningRate);
    }

    err = hipDeviceSynchronize();
    CUDA_CHECK(err, "Sync Training Threads");
}

//Predictions only work while training on the gpu, otherwise the data is not copied to the gpu memory.
extern "C" __declspec(dllexport) float* Predict(float* data, float* prediction) {

    hipError_t err = hipMemcpy(gpu_allLayer[0]->NeuronValues, data, cpu_allLayer[0]->Size * sizeof(float), hipMemcpyHostToDevice);
    CUDA_CHECK(err, "Memcpy Inputs for Prediction");

    FeedForward();

    BaseLayer * outLayer = gpu_allLayer[allLayerCount - 1];
    hipMemcpy(prediction, outLayer->NeuronValues, sizeof(outLayer->Size) * sizeof(float), hipMemcpyDeviceToHost);
}

extern "C" __declspec(dllexport) void Init(int totalLayers) {
    printf("Training on CUDA is enabled\n");
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    allLayerCount = totalLayers;

    // Allocate memory for cpu_allLayer array on the host
    cpu_allLayer = new BaseLayer * [totalLayers];
    gpu_allLayer = new BaseLayer * [totalLayers];

    // Initialize layers to null
    for (int i = 0; i < totalLayers; ++i) {
        cpu_allLayer[i] = nullptr;
        gpu_allLayer[i] = nullptr;
    }
}

extern "C" __declspec(dllexport) bool CheckCuda() {
    int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);

    return deviceCount >= 1;
}

void AllocateLayerMemory(BaseLayer* gpuLayer, BaseLayer* cpuLayer, int prevSize, int size, float* biases, float* weights, float* neuronValues, float* errors) {
    hipError_t err;

    // Assign host pointers
    cpuLayer->Weights = weights;
    cpuLayer->Biases = biases;
    cpuLayer->Errors = errors;
    cpuLayer->NeuronValues = neuronValues;

    // Allocate GPU memory and check for errors
    err = hipMalloc(&gpuLayer->Biases, size * sizeof(float));
    CUDA_CHECK(err, "Allocating GPU Biases");

    err = hipMalloc(&gpuLayer->NeuronValues, size * sizeof(float));
    CUDA_CHECK(err, "Allocating GPU NeuronValues");

    err = hipMalloc(&gpuLayer->Errors, size * sizeof(float));
    CUDA_CHECK(err, "Allocating GPU Errors");

    if (prevSize != 0) {
        err = hipMalloc(&gpuLayer->Weights, size * prevSize * sizeof(float));
        CUDA_CHECK(err, "Allocating GPU Weights");
    }
    
    // Copy host data to GPU
    err = hipMemcpy(gpuLayer->Biases, biases, size * sizeof(float), hipMemcpyHostToDevice);
    CUDA_CHECK(err, "Copying Biases to GPU");

    err = hipMemcpy(gpuLayer->NeuronValues, neuronValues, size * sizeof(float), hipMemcpyHostToDevice);
    CUDA_CHECK(err, "Copying NeuronValues to GPU");

    err = hipMemcpy(gpuLayer->Errors, errors, size * sizeof(float), hipMemcpyHostToDevice);
    CUDA_CHECK(err, "Copying Errors to GPU");

    if (prevSize != 0) {
        err = hipMemcpy(gpuLayer->Weights, weights, size * prevSize * sizeof(float), hipMemcpyHostToDevice);
        CUDA_CHECK(err, "Copying Weights to GPU");
    }

    //PrintLayerInfo(gpuLayer, "GPU Layer");
    //PrintLayerInfo(cpuLayer, "CPU Layer");
}

void InitNextLayers() {   
    //set the nextlayer for every layer, this needs to be done after all layers are initialized,
    //because otherwise, the pointers to the nextLayer do not exist yet
    
    for (int i = 0; i < allLayerCount; i++) {
        cpu_allLayer[i]->nextLayer = i + 1 > allLayerCount ? nullptr : cpu_allLayer[i + 1];
        gpu_allLayer[i]->nextLayer = i + 1 > allLayerCount ? nullptr : gpu_allLayer[i + 1];
    }
}

void FillLayer(BaseLayer* cpuLayer, BaseLayer* gpuLayer, int layerIndex, int size, int activation) {
    if (layerIndex == allLayerCount - 1) {
        InitNextLayers();
    }

    cpuLayer->Size = gpuLayer->Size = size;
    cpuLayer->Activation = gpuLayer->Activation = activation;

    cpuLayer->previousLayer = layerIndex == 0 ? nullptr : cpu_allLayer[layerIndex - 1];
    gpuLayer->previousLayer = layerIndex == 0 ? nullptr : gpu_allLayer[layerIndex - 1];
}

extern "C" __declspec(dllexport) void InitInputLayer(
    int layerIndex,
    int size,
    float* biases,
    float* weights,
    float* neuronValues,
    float* errors,
    int activation)
{

    gpu_allLayer[layerIndex] = new InputLayer();
    cpu_allLayer[layerIndex] = new InputLayer();

    BaseLayer* cpuLayer = cpu_allLayer[layerIndex];
    BaseLayer* gpuLayer = gpu_allLayer[layerIndex];

    FillLayer(cpuLayer, gpuLayer, layerIndex, size, activation);
    AllocateLayerMemory(gpuLayer, cpuLayer, 0, size, biases, weights, neuronValues, errors);
}

extern "C" __declspec(dllexport) void InitOutputLayer(
    int layerIndex,
    int prevSize,
    int size,
    float* biases,
    float* weights,
    float* neuronValues,
    float* errors,
    int activation)
{

    gpu_allLayer[layerIndex] = new OutputLayer();
    cpu_allLayer[layerIndex] = new OutputLayer();

    BaseLayer* cpuLayer = cpu_allLayer[layerIndex];
    BaseLayer* gpuLayer = gpu_allLayer[layerIndex];

    FillLayer(cpuLayer, gpuLayer, layerIndex, size, activation);
    AllocateLayerMemory(gpuLayer, cpuLayer, prevSize, size, biases, weights, neuronValues, errors);
}


extern "C" __declspec(dllexport) void InitDenseLayer(
    int layerIndex,
    int prevSize,
    int size,
    float* biases,
    float* weights,
    float* neuronValues,
    float* errors, 
    int activation) 
{
    gpu_allLayer[layerIndex] = new DenseLayer();
    cpu_allLayer[layerIndex] = new DenseLayer();

    BaseLayer* cpuLayer = cpu_allLayer[layerIndex];
    BaseLayer* gpuLayer = gpu_allLayer[layerIndex];

    FillLayer(cpuLayer, gpuLayer, layerIndex, size, activation);
    AllocateLayerMemory(gpuLayer, cpuLayer, prevSize, size, biases, weights, neuronValues, errors);
}

extern "C" __declspec(dllexport) void InitLSTMLayer(
    int layerIndex,
    int prevSize,
    int size,
    float* biases,
    float* neuronValues,
    float* errors,
    float* WeightsInput,
    float* WeightsForget,
    float* WeightsOutput,
    float* WeightsCandidate,
    float* CellState,
    float* OutputGate,
    float* ForgetGate,
    float* InputGate,
    float* CandidateCellState,
    float* inputGateGradients,
    float* forgetGateGradients,
    float* outputGradients,
    float* candidateCellGradients
)
{
    LSTMLayer* cpu_layer = new LSTMLayer();
    LSTMLayer* gpu_layer = new LSTMLayer();

    gpu_allLayer[layerIndex] = gpu_layer;
    cpu_allLayer[layerIndex] = cpu_layer;

    FillLayer(cpu_layer, gpu_layer, layerIndex, size, 0);
    AllocateLayerMemory(gpu_layer, cpu_layer, 0, size, biases, 0, neuronValues, errors);

    //allocate lstm memory:

    //all weights
    hipError_t err = hipMalloc(&gpu_layer->WeightsInput, size * prevSize * sizeof(float));
    CUDA_CHECK(err, "Allocating GPU WeightsInput");
    err = hipMemcpy(gpu_layer->WeightsInput, WeightsInput, size * prevSize * sizeof(float), hipMemcpyHostToDevice);
    CUDA_CHECK(err, "Copying WeightsInput to GPU");

    err = hipMalloc(&gpu_layer->WeightsForget, size * prevSize * sizeof(float));
    CUDA_CHECK(err, "Allocating GPU WeightsForget");
    err = hipMemcpy(gpu_layer->WeightsForget, WeightsForget, size * prevSize * sizeof(float), hipMemcpyHostToDevice);
    CUDA_CHECK(err, "Copying WeightsForget to GPU");

    err = hipMalloc(&gpu_layer->WeightsOutput, size * prevSize * sizeof(float));
    CUDA_CHECK(err, "Allocating GPU WeightsOutput");
    err = hipMemcpy(gpu_layer->WeightsOutput, WeightsOutput, size * prevSize * sizeof(float), hipMemcpyHostToDevice);
    CUDA_CHECK(err, "Copying WeightsOutput to GPU");

    err = hipMalloc(&gpu_layer->WeightsCandidate, size * prevSize * sizeof(float));
    CUDA_CHECK(err, "Allocating GPU WeightsCandidate");
    err = hipMemcpy(gpu_layer->WeightsCandidate, WeightsCandidate, size * prevSize * sizeof(float), hipMemcpyHostToDevice);
    CUDA_CHECK(err, "Copying WeightsCandidate to GPU");

    //gradients:
    err = hipMalloc(&gpu_layer->inputGateGradients, size * sizeof(float));
    CUDA_CHECK(err, "Allocating GPU inputGateGradients");
    err = hipMemcpy(gpu_layer->inputGateGradients, inputGateGradients, size * sizeof(float), hipMemcpyHostToDevice);
    CUDA_CHECK(err, "Copying inputGateGradients to GPU");
    
    err = hipMalloc(&gpu_layer->forgetGateGradients, size * sizeof(float));
    CUDA_CHECK(err, "Allocating GPU forgetGateGradients");
    err = hipMemcpy(gpu_layer->forgetGateGradients, forgetGateGradients, size * sizeof(float), hipMemcpyHostToDevice);
    CUDA_CHECK(err, "Copying forgetGateGradients to GPU");

    err = hipMalloc(&gpu_layer->outputGradients, size * sizeof(float));
    CUDA_CHECK(err, "Allocating GPU outputGradients");
    err = hipMemcpy(gpu_layer->outputGradients, outputGradients, size * sizeof(float), hipMemcpyHostToDevice);
    CUDA_CHECK(err, "Copying outputGradients to GPU");

    err = hipMalloc(&gpu_layer->candidateCellGradients, size * sizeof(float));
    CUDA_CHECK(err, "Allocating GPU candidateCellGradients");
    err = hipMemcpy(gpu_layer->candidateCellGradients, candidateCellGradients, size * sizeof(float), hipMemcpyHostToDevice);
    CUDA_CHECK(err, "Copying candidateCellGradients to GPU");

    err = hipMalloc(&gpu_layer->CellState, size * sizeof(float));
    CUDA_CHECK(err, "Allocating GPU CellState");
    err = hipMemcpy(gpu_layer->CellState, CellState, size * sizeof(float), hipMemcpyHostToDevice);
    CUDA_CHECK(err, "Copying CellState to GPU");

    err = hipMalloc(&gpu_layer->OutputGate, size * sizeof(float));
    CUDA_CHECK(err, "Allocating GPU OutputGate");
    err = hipMemcpy(gpu_layer->OutputGate, OutputGate, size * sizeof(float), hipMemcpyHostToDevice);
    CUDA_CHECK(err, "Copying OutputGate to GPU");

    err = hipMalloc(&gpu_layer->ForgetGate, size * sizeof(float));
    CUDA_CHECK(err, "Allocating GPU ForgetGate");
    err = hipMemcpy(gpu_layer->ForgetGate, ForgetGate, size * sizeof(float), hipMemcpyHostToDevice);
    CUDA_CHECK(err, "Copying ForgetGate to GPU");

    err = hipMalloc(&gpu_layer->InputGate, size * sizeof(float));
    CUDA_CHECK(err, "Allocating GPU InputGate");
    err = hipMemcpy(gpu_layer->InputGate, InputGate, size * sizeof(float), hipMemcpyHostToDevice);
    CUDA_CHECK(err, "Copying InputGate to GPU");
}

extern "C" __declspec(dllexport) void DoneLSTMLayer(int index) {
    hipError_t err;

    LSTMLayer* gpuLayer = dynamic_cast<LSTMLayer*>(gpu_allLayer[index]);
    LSTMLayer* cpuLayer = dynamic_cast<LSTMLayer*>(cpu_allLayer[index]);

    unsigned long long size = gpuLayer->Size* gpuLayer->previousLayer->Size * sizeof(float);

    err = hipMemcpy(cpuLayer->WeightsCandidate, gpuLayer->WeightsCandidate, size, hipMemcpyDeviceToHost);
    CUDA_CHECK(err, "10");
    err = hipMemcpy(cpuLayer->WeightsForget, gpuLayer->WeightsForget, size, hipMemcpyDeviceToHost);
    CUDA_CHECK(err, "11");
    err = hipMemcpy(cpuLayer->WeightsInput, gpuLayer->WeightsInput, size, hipMemcpyDeviceToHost);
    CUDA_CHECK(err, "12");
    err = hipMemcpy(cpuLayer->WeightsOutput, gpuLayer->WeightsOutput, size, hipMemcpyDeviceToHost);
    CUDA_CHECK(err, "12");

    hipFree(gpuLayer->WeightsCandidate);
    hipFree(gpuLayer->WeightsForget);
    hipFree(gpuLayer->WeightsInput);
    hipFree(gpuLayer->WeightsOutput);
    hipFree(gpuLayer->CellState);
    hipFree(gpuLayer->OutputGate);
    hipFree(gpuLayer->ForgetGate);
    hipFree(gpuLayer->InputGate);
    hipFree(gpuLayer->CandidateCellState);
    hipFree(gpuLayer->inputGateGradients);
    hipFree(gpuLayer->forgetGateGradients);
    hipFree(gpuLayer->outputGradients);
    hipFree(gpuLayer->candidateCellGradients);
}

extern "C" __declspec(dllexport) void Cleanup() {
    //free the memory of every layer from the gpu
    for (int i = 0; i < allLayerCount; i++) {
        BaseLayer* gpuLayer = gpu_allLayer[i];

        if (gpuLayer) {
            hipFree(gpuLayer->Biases);
            hipFree(gpuLayer->NeuronValues);
            hipFree(gpuLayer->Errors);
            hipFree(gpuLayer->Weights);
        }
    }

    hipFree(gpu_desiredValues);
    delete[] gpu_allLayer;
    //delete[] cpu_allLayer;
}

extern "C" __declspec(dllexport) void DoneTraining() {
    hipError_t err;

    // Copy updated weights and biases back to the host
    for (int i = 0; i < allLayerCount; i++) {
        BaseLayer* gpuLayer = gpu_allLayer[i];
        BaseLayer* cpuLayer = cpu_allLayer[i];

        //copy back all layers:
        err = hipMemcpy(cpuLayer->Biases, gpuLayer->Biases, gpuLayer->Size * sizeof(float), hipMemcpyDeviceToHost);
        CUDA_CHECK(err, "10");
        err = hipMemcpy(cpuLayer->Errors, gpuLayer->NeuronValues, gpuLayer->Size * sizeof(float), hipMemcpyDeviceToHost);
        CUDA_CHECK(err, "11");
        err = hipMemcpy(cpuLayer->NeuronValues, gpuLayer->NeuronValues, gpuLayer->Size * sizeof(float), hipMemcpyDeviceToHost);
        CUDA_CHECK(err, "12");

        //input layer doesn't have any weights:
        if (i != 0) {
            err = hipMemcpy(cpuLayer->Weights, gpuLayer->Weights, gpuLayer->Size * gpu_allLayer[i - 1]->Size * sizeof(float), hipMemcpyDeviceToHost);
            CUDA_CHECK(err, "9");

#ifdef DEBUG
            printf("SIZE: (%d), %d\n", i, (gpuLayer->Size * gpu_allLayer[i - 1]->Size));
#endif // DEBUG
        }

        //finish LSTM layer:
        if (LSTMLayer* d1 = dynamic_cast<LSTMLayer*>(gpuLayer))
            DoneLSTMLayer(i);

#ifdef DEBUG
        printf("MemoryAdress: %d, %p, %p, %p, %p\n", i, cpuLayer->Weights, cpuLayer->Biases, cpuLayer->Errors, cpuLayer->NeuronValues);
#endif // DEBUG
    }
    Cleanup();
    printf("\nCUDA: Done -> Cleaned Up Memory\n");
}
