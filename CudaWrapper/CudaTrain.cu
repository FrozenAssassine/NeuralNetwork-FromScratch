/*
* CUDA training code for my neural network!
* Variable declarations starting with gpu are allocated in gpu memory
*/

#include <iostream>

#include "hip/hip_runtime.h"
#include ""

#include "DenseLayer.h"
#include "InputLayer.h"
#include "OutputLayer.h"
#include "BaseLayer.h"

#include "StopWatch.h"
#include "AccuracyCalculator.h"

#define DEBUG

#define CUDA_CHECK(err, code) \
    if (err != hipSuccess) { \
        printf("\nCUDA: Error (%s): %s at line %d\n", code, hipGetErrorString(err), __LINE__); \
        exit(EXIT_FAILURE);\
    }

using namespace std;

const int threadsPerBlock = 32;

BaseLayer** gpu_allLayer = nullptr;
BaseLayer** cpu_allLayer = nullptr;
int allLayerCount;

float * gpu_desiredValues;


char* RepeatString(char s, int n) {
    char* outs = new char[n];
    for (int i = 0; i < n; i++) {
        outs[i] = s;
    }
    return outs;
}


void PrintLayerInfo(BaseLayer * layer, const char* layerName) {
    printf("%s - Weights: %p, Biases: %p, NeuronValues: %p, Errors: %p, Size: %d\n",
        layerName, layer->Weights, layer->Biases, layer->NeuronValues, layer->Errors, layer->Size);
}

void FeedForward() {
    hipError_t err;

    //first layer can be skipped -> feed forward all hidden and output:
    for (int i = 1; i < allLayerCount; i++) {
        gpu_allLayer[i]->FeedForward(threadsPerBlock);
    }

    err = hipDeviceSynchronize();
    CUDA_CHECK(err, "Feed Forward Synchronize Threads");
}

extern "C" __declspec(dllexport) void TrainSingle(float* inputs, float* desired, int size, float learningRate) {
    hipError_t err;

    //copy the next inputs & outputs to the gpu memory
    err = hipMemcpy(gpu_allLayer[0]->NeuronValues, inputs, cpu_allLayer[0]->Size * sizeof(float), hipMemcpyHostToDevice);
    CUDA_CHECK(err, "Memcpy Inputs");
    
    if (gpu_desiredValues == nullptr) {
        err = hipMalloc(&gpu_desiredValues, cpu_allLayer[allLayerCount - 1]->Size * sizeof(float));
        CUDA_CHECK(err, "Malloc Desired Values");
    }
    err = hipMemcpy(gpu_desiredValues, desired, cpu_allLayer[allLayerCount - 1]->Size * sizeof(float), hipMemcpyHostToDevice);
    CUDA_CHECK(err, "Memcopy Desired Values");

    FeedForward();
    
    for (int i = allLayerCount - 1; i >= 0; i--) {
        gpu_allLayer[i]->Train(threadsPerBlock, gpu_desiredValues, learningRate);
    }

    err = hipDeviceSynchronize();
    CUDA_CHECK(err, "Sync Training Threads");
}

extern "C" __declspec(dllexport) void Predict(float* data, float* prediction) {

    hipError_t err = hipMemcpy(gpu_allLayer[0]->NeuronValues, data, cpu_allLayer[0]->Size * sizeof(float), hipMemcpyHostToDevice);
    CUDA_CHECK(err, "Memcpy Inputs for Prediction");

    FeedForward();

    hipMemcpy(prediction, gpu_allLayer[allLayerCount - 1]->NeuronValues, cpu_allLayer[allLayerCount - 1]->Size * sizeof(float), hipMemcpyDeviceToHost);
}

extern "C" __declspec(dllexport) void Init(int totalLayers) {
    printf("Training on CUDA is enabled\n");
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    allLayerCount = totalLayers;

    // Allocate memory for cpu_allLayer array on the host
    cpu_allLayer = new BaseLayer * [totalLayers];
    gpu_allLayer = new BaseLayer * [totalLayers];

    // Initialize layers to null
    for (int i = 0; i < totalLayers; ++i) {
        cpu_allLayer[i] = nullptr;
        gpu_allLayer[i] = nullptr;
    }
}

extern "C" __declspec(dllexport) bool CheckCuda() {
    int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);

    return deviceCount >= 1;
}

void AllocateLayerMemory(BaseLayer* gpuLayer, BaseLayer* cpuLayer, int prevSize, int size, float* biases, float* weights, float* neuronValues, float* errors) {
    hipError_t err;

    // Assign host pointers
    cpuLayer->Weights = weights;
    cpuLayer->Biases = biases;
    cpuLayer->Errors = errors;
    cpuLayer->NeuronValues = neuronValues;

    // Allocate GPU memory and check for errors
    err = hipMalloc(&gpuLayer->Biases, size * sizeof(float));
    CUDA_CHECK(err, "Allocating GPU Biases");

    err = hipMalloc(&gpuLayer->NeuronValues, size * sizeof(float));
    CUDA_CHECK(err, "Allocating GPU NeuronValues");

    err = hipMalloc(&gpuLayer->Errors, size * sizeof(float));
    CUDA_CHECK(err, "Allocating GPU Errors");

    if (prevSize != 0) {
        err = hipMalloc(&gpuLayer->Weights, size * prevSize * sizeof(float));
        CUDA_CHECK(err, "Allocating GPU Weights");
    }
    
    // Copy host data to GPU
    err = hipMemcpy(gpuLayer->Biases, biases, size * sizeof(float), hipMemcpyHostToDevice);
    CUDA_CHECK(err, "Copying Biases to GPU");

    err = hipMemcpy(gpuLayer->NeuronValues, neuronValues, size * sizeof(float), hipMemcpyHostToDevice);
    CUDA_CHECK(err, "Copying NeuronValues to GPU");

    err = hipMemcpy(gpuLayer->Errors, errors, size * sizeof(float), hipMemcpyHostToDevice);
    CUDA_CHECK(err, "Copying Errors to GPU");

    if (prevSize != 0) {
        err = hipMemcpy(gpuLayer->Weights, weights, size * prevSize * sizeof(float), hipMemcpyHostToDevice);
        CUDA_CHECK(err, "Copying Weights to GPU");
    }

    //PrintLayerInfo(gpuLayer, "GPU Layer");
    //PrintLayerInfo(cpuLayer, "CPU Layer");
}

void InitNextLayers() {   
    //set the nextlayer for every layer, this needs to be done after all layers are initialized,
    //because otherwise, the pointers to the nextLayer do not exist yet
    
    for (int i = 0; i < allLayerCount; i++) {
        cpu_allLayer[i]->nextLayer = i + 1 > allLayerCount ? nullptr : cpu_allLayer[i + 1];
        gpu_allLayer[i]->nextLayer = i + 1 > allLayerCount ? nullptr : gpu_allLayer[i + 1];
    }
}

void FillLayer(BaseLayer* cpuLayer, BaseLayer* gpuLayer, int layerIndex, int size, int activation) {
    if (layerIndex == allLayerCount - 1) {
        InitNextLayers();
    }

    cpuLayer->Size = gpuLayer->Size = size;
    cpuLayer->Activation = gpuLayer->Activation = activation;

    cpuLayer->previousLayer = layerIndex == 0 ? nullptr : cpu_allLayer[layerIndex - 1];
    gpuLayer->previousLayer = layerIndex == 0 ? nullptr : gpu_allLayer[layerIndex - 1];
}

extern "C" __declspec(dllexport) void InitInputLayer(
    int layerIndex,
    int size,
    float* biases,
    float* weights,
    float* neuronValues,
    float* errors,
    int activation)
{

    gpu_allLayer[layerIndex] = new InputLayer();
    cpu_allLayer[layerIndex] = new InputLayer();

    BaseLayer* cpuLayer = cpu_allLayer[layerIndex];
    BaseLayer* gpuLayer = gpu_allLayer[layerIndex];

    FillLayer(cpuLayer, gpuLayer, layerIndex, size, activation);
    AllocateLayerMemory(gpuLayer, cpuLayer, 0, size, biases, weights, neuronValues, errors);
}

extern "C" __declspec(dllexport) void InitOutputLayer(
    int layerIndex,
    int prevSize,
    int size,
    float* biases,
    float* weights,
    float* neuronValues,
    float* errors,
    int activation)
{

    gpu_allLayer[layerIndex] = new OutputLayer();
    cpu_allLayer[layerIndex] = new OutputLayer();

    BaseLayer* cpuLayer = cpu_allLayer[layerIndex];
    BaseLayer* gpuLayer = gpu_allLayer[layerIndex];

    FillLayer(cpuLayer, gpuLayer, layerIndex, size, activation);
    AllocateLayerMemory(gpuLayer, cpuLayer, prevSize, size, biases, weights, neuronValues, errors);
}

extern "C" __declspec(dllexport) void TrainFull(
    float* inputX, 
    float* desired, 
    int epochs, 
    int samples, //number of items to train
    int features, //number of items per input item because 2d array is flattened
    int outputs, //number of items per desired item because 2d array is flattened
    float learningRate = 0.1f,
    int loggingInterval = 100,
    int epochInterval = 1, 
    float evaluatePercent = 10)
{
    Stopwatch epochTime = new Stopwatch(true);
    Stopwatch stepTime = new Stopwatch(true);
    AccuracyCalculator accCalc;

    for (int e = 0; e < epochs; e++) {

        epochTime.Start();
        stepTime.Start();
        accCalc.NextEpoch();

        float averageStepTime = 0;

        for (int i = 0; i < samples; i++) {
            float* x = &inputX[i * features];
            float* y = &desired[i * outputs];

            TrainSingle(x, y, features, 0.01f);

            //use this, when each epoch takes longer due to more items that need to compute
            if ((i + 1) % loggingInterval == 0)
            {
                stepTime.Stop();

                averageStepTime += stepTime.ElapsedMilliseconds();
                printf("Epoch %d/%d; %d/%d; (%.3fms)\n", e + 1, epochs, i + 1, samples, stepTime.ElapsedMilliseconds());
                stepTime.Start();
            }
        }

        //print epoch every x epochs (default: 100) => for fast training
        if ((e + 1) % epochInterval == 0)
        {
            accCalc.Calculate(inputX, desired, &Predict, samples, features, outputs, 0);

            printf("%s\n", RepeatString('-', 50));
            printf("Epoch %d took %.3fms;", e + 1, epochTime.ElapsedMilliseconds());
            accCalc.PrintAccuracy();
            if (averageStepTime > 0) {
                printf(" avg(%d)ms/step", (int)averageStepTime / (samples / loggingInterval));
            }
            printf("\n");

            //dont print the last line after training => looks weird :D
            if (e != epochs - 1)
                printf("%s\n", RepeatString('-', 50));
        }
    }
}

extern "C" __declspec(dllexport) void InitDenseLayer(
    int layerIndex,
    int prevSize,
    int size,
    float* biases,
    float* weights,
    float* neuronValues,
    float* errors, 
    int activation) 
{
    gpu_allLayer[layerIndex] = new DenseLayer();
    cpu_allLayer[layerIndex] = new DenseLayer();

    BaseLayer* cpuLayer = cpu_allLayer[layerIndex];
    BaseLayer* gpuLayer = gpu_allLayer[layerIndex];

    FillLayer(cpuLayer, gpuLayer, layerIndex, size, activation);
    AllocateLayerMemory(gpuLayer, cpuLayer, prevSize, size, biases, weights, neuronValues, errors);
}


extern "C" __declspec(dllexport) void Cleanup() {
    //free the memory of every layer from the gpu
    for (int i = 0; i < allLayerCount; i++) {
        BaseLayer* gpuLayer = gpu_allLayer[i];

        if (gpuLayer) {
            hipFree(gpuLayer->Biases);
            hipFree(gpuLayer->NeuronValues);
            hipFree(gpuLayer->Errors);
            hipFree(gpuLayer->Weights);
        }
    }

    hipFree(gpu_desiredValues);
    delete[] gpu_allLayer;
    //delete[] cpu_allLayer;

    printf("\nCUDA: Done -> Cleaned Up Memory\n");
}

extern "C" __declspec(dllexport) void DoneTraining() {
    hipError_t err;

    // Copy updated weights and biases back to the host
    for (int i = 0; i < allLayerCount; i++) {
        BaseLayer* gpuLayer = gpu_allLayer[i];
        BaseLayer* cpuLayer = cpu_allLayer[i];

        //copy back all layers:
        err = hipMemcpy(cpuLayer->Biases, gpuLayer->Biases, gpuLayer->Size * sizeof(float), hipMemcpyDeviceToHost);
        CUDA_CHECK(err, "10");
        err = hipMemcpy(cpuLayer->Errors, gpuLayer->NeuronValues, gpuLayer->Size * sizeof(float), hipMemcpyDeviceToHost);
        CUDA_CHECK(err, "11");
        err = hipMemcpy(cpuLayer->NeuronValues, gpuLayer->NeuronValues, gpuLayer->Size * sizeof(float), hipMemcpyDeviceToHost);
        CUDA_CHECK(err, "12");

        //input layer doesn't have any weights:
        if (i != 0) {
            err = hipMemcpy(cpuLayer->Weights, gpuLayer->Weights, gpuLayer->Size * gpu_allLayer[i - 1]->Size * sizeof(float), hipMemcpyDeviceToHost);
            CUDA_CHECK(err, "9");

#ifdef DEBUG
            printf("SIZE: (%d), %d\n", i, (gpuLayer->Size * gpu_allLayer[i - 1]->Size));
#endif // DEBUG
        }
#ifdef DEBUG
        printf("MemoryAdress: %d, %p, %p, %p, %p\n", i, cpuLayer->Weights, cpuLayer->Biases, cpuLayer->Errors, cpuLayer->NeuronValues);
#endif // DEBUG
    }
}