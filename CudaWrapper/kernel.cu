#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>

#include <chrono>
#include <stdio.h>
#include <iostream>
#include <vector>

//show more ouputs like memory adress etc.
//#define DEBUG

int threadsPerBlock = 1024;

typedef struct Layer {
    float* Biases;
    float* NeuronValues;
    float* Errors;
    float* Weights;
    int Size;
    int Activation;
} Layer;

int allLayersCount;
Layer* gpuLayers;
Layer* cpuLayers;

float* desiredValues;

#define CUDA_CHECK(err, code) \
    if (err != hipSuccess) { \
        printf("\nCUDA: Error (%s): %s at line %d\n", code, hipGetErrorString(err), __LINE__); \
    }

__device__ float Activation(float x, int activation) {
    switch (activation) {
    case 0: //sigmoid
        return 1.0f / (1.0f + expf(-x));
    case 1: //relu
        return fmaxf(0.0f, x);
    case 2: //softmax
        return expf(x) / (1.0f + expf(x));
    }
}

__device__ float ActivationDeriv(float x, int activation) {
    switch (activation) {
    case 0: //sigmoid deriv
        return x * (1 - x);
    case 1: //relu deriv
        return x > 0.0f ? 1.0f : 0.0f;
    case 2: //softmax deriv
        return x * (1.0f - x);
    }
}

__global__ void ff_hiddenValues(Layer current, Layer previous, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float sum = 0.0f;
        int index = idx * previous.Size;
        for (int j = 0; j < previous.Size; j++) {
            sum += previous.NeuronValues[j] * current.Weights[index + j];
        }
        current.NeuronValues[idx] = Activation(sum + current.Biases[idx], current.Activation);
    }
}

__global__ void ff_outputValues(Layer output, Layer prev, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float sum = 0.0f;
        int weightIndex = idx * prev.Size;
        for (int j = 0; j < prev.Size; j++) {
            sum += prev.NeuronValues[j] * output.Weights[weightIndex + j];
        }
        output.NeuronValues[idx] = Activation(sum + output.Biases[idx], output.Activation);
    }
}

__global__ void output_WeightsBiases(Layer output, Layer previous, float learningRate, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float derivNeuronVal = learningRate * output.Errors[idx] * ActivationDeriv(output.NeuronValues[idx], output.Activation);
        int weightIndex = idx * previous.Size;

        for (int j = 0; j < previous.Size; j++) {
            atomicAdd(&output.Weights[weightIndex + j], derivNeuronVal * previous.NeuronValues[j]);
        }
        atomicAdd(&output.Biases[idx], learningRate * output.Errors[idx] * ActivationDeriv(output.NeuronValues[idx], output.Activation));
    }
}

__global__ void hidden_ErrorWeight(Layer next, Layer prev, Layer cur, float learningRate, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float err = 0.0f;
        int index = idx * prev.Size;

        for (int j = 0; j < next.Size; j++) {
            err += (next.Errors[j] * next.Weights[j * cur.Size + idx]);
        }
        float error = err * ActivationDeriv(cur.NeuronValues[idx], cur.Activation);
        cur.Errors[idx] = error;

        error *= learningRate;

        for (int j = 0; j < prev.Size; j++) {
            atomicAdd(&cur.Weights[index + j], error * prev.NeuronValues[j]);
        }
        atomicAdd(&cur.Biases[idx], error);
    }
}

__global__ void output_Errors(Layer output, float* desired, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output.Errors[idx] = desired[idx] - output.NeuronValues[idx];
    }
}

float* FeedForward(float* data, int n) {
    hipError_t err;

    //calculate neuron values for hidden layer:
    for (int i = 1; i < allLayersCount - 1; i++) {
        Layer& prevLayer = gpuLayers[i - 1];
        Layer& currLayer = gpuLayers[i];

        int blocks = (currLayer.Size + threadsPerBlock - 1) / threadsPerBlock;
        ff_hiddenValues<< < blocks, threadsPerBlock >> > (currLayer, prevLayer, currLayer.Size);
    }

    //Compute neuron values for output layer
    Layer& prevLayer = gpuLayers[allLayersCount - 2];
    Layer& outLayer = gpuLayers[allLayersCount - 1];

    int blocks = (outLayer.Size + threadsPerBlock - 1) / threadsPerBlock;
    ff_outputValues<< <blocks, threadsPerBlock >> > (outLayer, prevLayer, outLayer.Size);

    hipDeviceSynchronize();
    err = hipGetLastError();
    CUDA_CHECK(err, "2");

    return outLayer.NeuronValues;
}

extern "C" __declspec(dllexport) void Train(float* inputs, float* desiredOutputs, int size, float learningRate) {
    hipError_t err;
    Layer& outputLayer = gpuLayers[allLayersCount - 1];
    Layer& prevLayer = gpuLayers[allLayersCount - 2];

    //copy the new inputs and outputs to the gpu
    err = hipMemcpy(gpuLayers[0].NeuronValues, inputs, cpuLayers[0].Size * sizeof(float), hipMemcpyHostToDevice);
    CUDA_CHECK(err, "3");

    if (desiredValues == nullptr) {
        err = hipMalloc(&desiredValues, cpuLayers[allLayersCount - 1].Size * sizeof(float));
        CUDA_CHECK(err, "50");
    }
    err = hipMemcpy(desiredValues, desiredOutputs, cpuLayers[allLayersCount - 1].Size * sizeof(float), hipMemcpyHostToDevice);
    CUDA_CHECK(err, "4");

    // Perform feedforward pass to get the network's output
    FeedForward(gpuLayers[0].NeuronValues, size);

    // Calculate errors for the output layer
    int outputBlocks = (outputLayer.Size + threadsPerBlock - 1) / threadsPerBlock;
    output_Errors << <outputBlocks, threadsPerBlock >> > (gpuLayers[allLayersCount - 1], desiredValues, outputLayer.Size);

    // Update weights and biases for the output layer
    output_WeightsBiases<< <outputBlocks, threadsPerBlock >> > (gpuLayers[allLayersCount - 1], gpuLayers[allLayersCount - 2], learningRate, outputLayer.Size);

    // Backpropagate the errors to the hidden layers
    for (int i = allLayersCount - 2; i >= 1; i--) {
        Layer& currLayer = gpuLayers[i];
        Layer& prevLayer = gpuLayers[i - 1];
        Layer& nextLayer = gpuLayers[i + 1];

        int errorBlocks = (currLayer.Size + threadsPerBlock - 1) / threadsPerBlock;
        hidden_ErrorWeight<< <errorBlocks, threadsPerBlock>> > (nextLayer, prevLayer, currLayer, learningRate, currLayer.Size);
    }

    hipDeviceSynchronize();
    err = hipGetLastError();
    CUDA_CHECK(err, "7");
}

extern "C" __declspec(dllexport) void Cleanup() {
    //free the memory of every layer from the gpu
    for (int i = 0; i < allLayersCount; i++) {
        hipFree(gpuLayers[i].Biases);
        hipFree(gpuLayers[i].NeuronValues);
        hipFree(gpuLayers[i].Errors);
        hipFree(gpuLayers[i].Weights);
    }

    hipFree(desiredValues);
    delete[] gpuLayers;
}

extern "C" __declspec(dllexport) void DoneTraining() {
    hipError_t err;

    // Copy updated weights and biases back to the host
    for (int i = 0; i < allLayersCount; i++) {
        Layer& gpuLayer = gpuLayers[i];
        Layer& cpuLayer = cpuLayers[i];
        
        //copy back all layers:
        err = hipMemcpy(cpuLayer.Biases, gpuLayer.Biases, gpuLayer.Size * sizeof(float), hipMemcpyDeviceToHost);
        CUDA_CHECK(err, "10");
        err = hipMemcpy(cpuLayer.Errors, gpuLayer.NeuronValues, gpuLayer.Size * sizeof(float), hipMemcpyDeviceToHost);
        CUDA_CHECK(err, "11");
        err = hipMemcpy(cpuLayer.NeuronValues, gpuLayer.NeuronValues, gpuLayer.Size * sizeof(float), hipMemcpyDeviceToHost);
        CUDA_CHECK(err, "12");

        //input layer doesn't have any weights:
        if (i != 0) {
            err = hipMemcpy(cpuLayer.Weights, gpuLayer.Weights, gpuLayer.Size * gpuLayers[i - 1].Size * sizeof(float), hipMemcpyDeviceToHost);
            CUDA_CHECK(err, "9");
            
#ifdef DEBUG
            printf("SIZE: (%d), %d\n", i, (gpuLayer.Size * gpuLayers[i - 1].Size));
#endif // DEBUG
        }
#ifdef DEBUG
        printf("MemoryAdress: %d, %p, %p, %p, %p\n", i, cpuLayer.Weights, cpuLayer.Biases, cpuLayer.Errors, cpuLayer.NeuronValues);
#endif // DEBUG
    }
    Cleanup();
    printf("\nCUDA: Done -> Cleaned Up Memory\n");
}

extern "C" __declspec(dllexport) bool CheckCuda() {
    int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);

    return deviceCount >= 1;
}

extern "C" __declspec(dllexport) void Init(int totalLayer) {
    printf("Training on CUDA is enabled\n");
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    threadsPerBlock = prop.maxThreadsPerBlock;

    allLayersCount = totalLayer;

    gpuLayers = new Layer[totalLayer];
    cpuLayers = new Layer[totalLayer];
}

extern "C" __declspec(dllexport) void InitLayer(int layerIndex, int prevSize, int size, float* biases, float* weights, float* values, float* errors, int activation) {
#ifdef DEBUG
    printf("Initializing layer %d with size %d and prevSize %d\n", layerIndex, size, prevSize);
#endif // DEBUG

    Layer& gpuLayer = gpuLayers[layerIndex];
    gpuLayer.Size = size;
    gpuLayer.Activation = activation;

    Layer& cpuLayer = cpuLayers[layerIndex];
    cpuLayer.Size = size;
    cpuLayer.Biases = biases;
    cpuLayer.Errors = errors;
    cpuLayer.NeuronValues = values;
    cpuLayer.Weights = weights;
    cpuLayer.Activation = activation;

#ifdef DEBUG:

    // Print the host pointers to verify they are valid
    printf("Host Biases: %p\n", biases);
    printf("Host Weights: %p\n", weights);
    printf("Host NeuronValues: %p\n", values);
    printf("Host Errors: %p\n", errors);
#endif

    // Allocate GPU memory
    hipError_t err;
    err = hipMalloc(&gpuLayer.Biases, size * sizeof(float));
    CUDA_CHECK(err, "25");
    err = hipMalloc(&gpuLayer.NeuronValues, size * sizeof(float));
    CUDA_CHECK(err, "26");
    err = hipMalloc(&gpuLayer.Errors, size * sizeof(float));
    CUDA_CHECK(err, "27");

    if (prevSize != 0) {
        err = hipMalloc(&gpuLayer.Weights, size * prevSize * sizeof(float));
        CUDA_CHECK(err, "28");
    }

#ifdef DEBUG:
    // Print the GPU pointers to ensure they are allocated
    printf("GPU Biases: %p\n", gpuLayer.Biases);
    printf("GPU NeuronValues: %p\n", gpuLayer.NeuronValues);
    printf("GPU Errors: %p\n", gpuLayer.Errors);
    printf("GPU Weights: %p\n", gpuLayer.Weights);
#endif

    // Copy initial data to GPU
    err = hipMemcpy(gpuLayer.Biases, cpuLayer.Biases, size * sizeof(float), hipMemcpyHostToDevice);
    CUDA_CHECK(err, "29");
    err = hipMemcpy(gpuLayer.NeuronValues, cpuLayer.NeuronValues, size * sizeof(float), hipMemcpyHostToDevice);
    CUDA_CHECK(err, "30");
    err = hipMemcpy(gpuLayer.Errors, cpuLayer.Errors, size * sizeof(float), hipMemcpyHostToDevice);
    CUDA_CHECK(err, "31");
    if (prevSize != 0) {
        err = hipMemcpy(gpuLayer.Weights, cpuLayer.Weights, size * prevSize * sizeof(float), hipMemcpyHostToDevice);
        CUDA_CHECK(err, "32");
    }

#ifdef DEBUG
    printf("Layer %d initialized successfully\n", layerIndex);
#endif // DEBUG
}
